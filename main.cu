#include "hip/hip_runtime.h"
#include <omp.h>
#include <iostream>
#include <string.h>
#include <fstream>
#include <algorithm>
#include "stdlib.h"
#include <stdio.h>
#include <math.h>
#include <vector>

#include "CSR_GRAPH.h"
#include "generator.h"
#include "device_gather.h"

//#include "/usr/local/cuda-10.1/include/hip/hip_runtime.h"
#include "hip/hip_runtime.h"

using namespace std;

#ifndef uint32_t
#define uint32_t int
#endif


int main(int argc, char **argv) {
    try {

        hipEvent_t start,stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);


        int threads = omp_get_max_threads();
        int vertices_index = atoi(argv[1]);
        int density_degree = atoi(argv[2]);
        char *graph_type = argv[3];
        //double begin,end;

        int vertices_count =  pow(2.0, vertices_index);
        int edges_count = density_degree * vertices_count;

        int *src_ids = new int[edges_count];
        int *dst_ids = new int[edges_count];
        float *weights = new float[edges_count];


        if (strcmp(graph_type, "rmat") == 0) {
            R_MAT(src_ids, dst_ids, weights, vertices_count, edges_count, 45, 20, 20, 15, threads, true, true);

        } else {
            uniform_random(src_ids, dst_ids, weights, vertices_count, edges_count, threads, true, true);
        }

        /*for (int i = 0; i < edges_count; i++) {
            cout << src_ids[i] << "----" << dst_ids[i] << endl;

        }*/

        CSR_GRAPH a(vertices_count,edges_count,src_ids,dst_ids,weights, true);


        //a.print_CSR_format();
        //a.print_adj_format();
        //a.adj_distribution(edges_count);



        a.generate_labels(threads);


        //begin = omp_get_wtime();
        //a.form_label_array(threads);
        //end = omp_get_wtime();

        a.move_to_device();


        hipEventRecord(start);

        dim3 block(1024,1);
        dim3 grid(vertices_count/block.x,1);

        device_gather <<<grid,block>>> (a.get_dev_v_array(),a.get_dev_e_array(),a.get_dev_dest_labels(),a.get_dev_labels());
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float time;
        hipEventElapsedTime(&time,start,stop);
        time*=1000;
        a.move_to_host();
        //a.print_label_info(threads);
        printf("Bandwidth for 2^%d edges is %f GB/s\n ", vertices_index + (int) log2((double)density_degree) , sizeof(int)*(vertices_count + 3*edges_count)/(time*(int)pow(1000,3)));

        /*begin = omp_get_wtime();
        a.form_label_array(threads);
        end = omp_get_wtime();
        //a.print_label_info(threads);
        printf("Time for 2^%d edges is %f\n ", vertices_index + (int) log2(density_degree) ,end - begin);

        begin = omp_get_wtime();
        a.form_label_array(threads);
        end = omp_get_wtime();
        //a.print_label_info(threads);
        printf("Time for 2^%d edges is %f\n ", vertices_index + (int) log2(density_degree) ,end - begin);
*/


        delete[] src_ids;
        delete[] dst_ids;
        delete[] weights;
    }
    catch (const char *error) {
        cout << error << endl;
        getchar();
        return 1;
    }
    catch (...) {
        cout << "unknown error" << endl;
    }
    return 0;
}