#include "hip/hip_runtime.h"
//#include "./moderngpu/src/moderngpu/kernel_segsort.hxx"
//#include "./moderngpu/src/moderngpu/memory.hxx"
//#include "./moderngpu/src/moderngpu/kernel_scan.hxx"
#include "./moderngpu/kernel_segsort.hxx"
#include "./moderngpu/memory.hxx"
#include "./moderngpu/kernel_scan.hxx"
#include <omp.h>
#include <iostream>
#include <string.h>
#include <fstream>
#include <algorithm>
#include "stdlib.h"
#include "CSR_GRAPH.h"
#include <stdio.h>
#include <math.h>
#include <sstream>
#include <string>
#include <vector>
#include "lp.h"
#include "CSR_GRAPH.h"
#include "generator.h"

//#include "./moderngpu/kernel_segsort.hxx"
//#include "./moderngpu/memory.hxx"
#include "device_gather.h"

//#include "/usr/local/cuda-10.1/include/hip/hip_runtime.h"
#include "hip/hip_runtime.h"
//#include "/usr/local/cuda-10.1/include/hip/hip_runtime_api.h"
#include "hip/hip_runtime_api.h"

#include "map"

#define SAFE_CALL(CallInstruction) { \
    hipError_t cuerr = CallInstruction; \
    if(cuerr != hipSuccess) { \
         printf("CUDA error: %s at call \"" #CallInstruction "\"\n", hipGetErrorString(cuerr)); \
         throw "error in CUDA API function, aborting..."; \
    } \
}

#define SAFE_KERNEL_CALL(KernelCallInstruction){ \
    KernelCallInstruction; \
    hipError_t cuerr = hipGetLastError(); \
    if(cuerr != hipSuccess) { \
        printf("CUDA error in kernel launch: %s at kernel \"" #KernelCallInstruction "\"\n", hipGetErrorString(cuerr)); \
        throw "error in CUDA kernel launch, aborting..."; \
    } \
    cuerr = hipDeviceSynchronize(); \
    if(cuerr != hipSuccess) { \
        printf("CUDA error in kernel execution: %s at kernel \"" #KernelCallInstruction "\"\n", hipGetErrorString(cuerr)); \
        throw "error in CUDA kernel execution, aborting..."; \
    } \
}


void label_stats(unsigned int *labels, unsigned int vertices_count) { // Почему то в map много нулей
    std::map<unsigned int, int> mp;
    for (unsigned int i = 0; i < vertices_count; i++) {
        if (mp.count(labels[i])) {
            mp[labels[i]]++;
        } else {
            mp[labels[i]] = 1;
        }
    }
    std::map<int, int> components;
    for (auto it = mp.begin(); it != mp.end(); it++) {
        if (components.count(it->second)) {
            components[it->second]++;
        } else {
            components[it->second] = 1;
        }
    }
    for (auto it = components.begin(); it != components.end(); it++) {
        if (it->first != 0) {
            cout << "there are " << it->second << " components of size " << it->first << endl;
        }
    }
}



void input(char *filename, bool directed, unsigned int *&src_ids, unsigned int *&dst_ids, unsigned int &vertices_count,
           unsigned int &edges_count) {
    unsigned int max_vertice = 0;

    std::ifstream infile(filename);
    std::string line;
    unsigned int i = 0;

    while (std::getline(infile, line)) {
        std::istringstream iss(line);
        int a, b;
        if (!(iss >> a >> b)) {
            break;
        } else {
            if (max(a, b) > max_vertice) {
                max_vertice = (unsigned) max(a, b);
            }
        }
        i++;
    }
    edges_count = i;
    vertices_count = max_vertice;
    src_ids = new unsigned int[edges_count];
    dst_ids = new unsigned int[edges_count];

    std::ifstream infile1(filename);
    i = 0;
    while (std::getline(infile1, line)) {
        std::istringstream iss(line);
        unsigned int a, b;
        if (!(iss >> a >> b)) {
            break;
        } else {
            src_ids[i] = a;
            dst_ids[i] = b;
            if (!directed) {
                src_ids[i + 1] = b;
                dst_ids[i + 1] = a;
            }
        }
        i++;
    }
}


__global__ void extract_boundaries_initial(bool *boundaries, unsigned int * v_array, unsigned int edges_count){

    unsigned long int i = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned long int position = v_array[i];
    if(i != 0){
        boundaries[position - 1] = 1;
    } else {
        boundaries[edges_count - 1] = 1;
    }
}

__global__ void extract_boundaries_optional(bool * boundaries, int *dest_labels, unsigned int edges_count){
    unsigned long int i = threadIdx.x + blockIdx.x * blockDim.x;
    if((boundaries[i] != 1)&&(i<edges_count)){
        if(dest_labels[i]!=dest_labels[i+1]){
            boundaries[i] = 1;
        }
    }
}

using namespace std;


int main(int argc, char **argv) {
    try {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        int threads = omp_get_max_threads();
        int vertices_index;
        int density_degree;
        bool check_flag = false;
        bool test_flag = false;
        char *graph_type;
        bool lp_flag = false;
        bool gather_flag = false;
        char *test_file = NULL;
        for (int i = 1; i < argc; i++) {
            string option(argv[i]);

            if ((option.compare("-scale") == 0) || (option.compare("-s") == 0)) {
                vertices_index = atoi(argv[++i]);
            }

            if ((option.compare("-edges") == 0) || (option.compare("-e") == 0)) {
                density_degree = atoi(argv[++i]);
            }

            if ((option.compare("-check") == 0)) {
                check_flag = true;
            }

            if ((option.compare("-nocheck") == 0)) {
                check_flag = false;
            }
            if ((option.compare("-type") == 0)) {
                graph_type = argv[++i];
            }
            if ((option.compare("-testing") == 0)) {
                test_file = argv[++i];
                test_flag = true;
                cout << "FLAG FOUND" << endl;
            }
            if ((option.compare("-lp")) == 0) {
                lp_flag = true;
            }
            if ((option.compare("-gather")) == 0) {
                gather_flag = true;
            }

        }

        unsigned int vertices_count = pow(2.0, vertices_index);
        unsigned int edges_count = density_degree * vertices_count;
        unsigned int *src_ids = NULL;
        unsigned int *dst_ids = NULL;
        float *weights = new float[edges_count];

        if (!test_flag) {
            src_ids = new unsigned int[edges_count];
            dst_ids = new unsigned int[edges_count];
            cout << "test_flag" << endl;
            if (strcmp(graph_type, "rmat") == 0) {
                R_MAT(src_ids, dst_ids, weights, vertices_count, edges_count, 45, 20, 20, 15, threads, true, true);

            } else {
                cout << "UR_GEN" << endl;
                uniform_random(src_ids, dst_ids, weights, vertices_count, edges_count, threads, true, true);
                cout << "Generated_UR" << endl;
            }
        } else {
            cout << test_flag << endl;
            cout << "file_init" << endl;
            input(test_file, false, src_ids, dst_ids, vertices_count, edges_count);
            vertices_count++;
            cout << "vertices:" << vertices_count << endl;
            cout << "edges: " << edges_count << endl;
        }


//        for (int i = 0; i < edges_count; i++) {
//            cout << src_ids[i] << "----" << dst_ids[i] << endl;
//        }

        cout << endl;
        CSR_GRAPH a(vertices_count, edges_count, src_ids, dst_ids, weights, true);
        a.save_to_graphviz_file("graph_pred", NULL);
        a.print_CSR_format();

        unsigned int *labels = new unsigned int[vertices_count];
        for (unsigned int j = 0; j < vertices_count; j++) {
            labels[j] = j;
        }
        cout<<"LABELS : ";
        for (unsigned int j = 0; j < vertices_count; j++) {
            cout<<labels[j]<<" ";
        }
        unsigned int *dest_labels = new unsigned int[edges_count];
        unsigned int *dev_labels;
        unsigned int *dev_dest_labels;
        bool *f_array;

        if (gather_flag) {

            SAFE_CALL((hipMalloc((void **) &dev_labels, (size_t) (sizeof(unsigned int)) * (vertices_count))));
            SAFE_CALL((hipMalloc((void **) &dev_dest_labels, (size_t) (sizeof(unsigned int)) * edges_count)));
            SAFE_CALL((hipMalloc((void **) &f_array, (size_t) (sizeof(bool)) * edges_count)));
            SAFE_CALL((hipMemset(f_array,0,(size_t) (sizeof(bool)) * edges_count)));

            a.move_to_device(dest_labels, labels, dev_dest_labels, dev_labels);

            SAFE_CALL(hipEventRecord(start));

            {
//                dim3 block(1024, 1);
//                dim3 grid(vertices_count * 32 / block.x, 1);
                  dim3 block(32*vertices_count,1);
                  dim3 grid(1,1); //only for test

                printf("starting...");
                SAFE_KERNEL_CALL((gather_warp_per_vertex << < grid, block >> >
                                                                    (a.get_dev_v_array(), a.get_dev_e_array(), dev_dest_labels, dev_labels, edges_count, vertices_count)));
            }
            printf("terminating....");
            SAFE_CALL(hipEventRecord(stop));
            SAFE_CALL(hipEventSynchronize(stop));
            float time;
            SAFE_CALL(hipEventElapsedTime(&time, start, stop));
            time *= 1000000;
            a.move_to_host(dest_labels, labels, dev_dest_labels, dev_labels);
            SAFE_CALL(hipFree(dev_labels));


            if (check_flag) {
                unsigned int *test_dest_labels = new unsigned int[edges_count];
                form_label_array(threads, vertices_count, edges_count, test_dest_labels, a.get_dev_v_array(), labels,
                                 a.get_e_array());
                int flag = check(edges_count, dest_labels, test_dest_labels);
                if (flag == 0) {
                    printf("CORRECT");
                }
                delete[] test_dest_labels;
            }


            printf("GATHER Bandwidth for 2^%d vertices and 2^%d edges is %f GB/s\n ", vertices_index,
                   vertices_index + (int) log2((double) density_degree),
                   sizeof(unsigned int) * (2 * vertices_count + 2 * edges_count) / (time));



            mgpu::standard_context_t context;
            std::vector<int> mem_gathered;
            for (int k = 0; k < edges_count; k++) {
                mem_gathered.push_back(dest_labels[k]);
            }
            std::vector<int> segs_host;
            for (int k = 0; k < vertices_count; k++) {
                segs_host.push_back(a.get_v_array()[k]);
            }

            for(int i = 0; i<segs_host.size();i++){
                if(i == 0){
                    std::cout<<"[ "<<0<<" , "<<segs_host[0] - 1<<" ]"<<std::endl;
                    std::cout<<"[ "<<segs_host[i]<<" , "<< segs_host[i+1] - 1 <<" ]"<<std::endl;
                    continue;
                }
                if(i == segs_host.size() - 1){
                    std::cout<<"[ "<<segs_host[segs_host.size() - 1 ]<<" , "<<  edges_count -1 <<" ]"<<std::endl; ;
                    continue;
                }
                std::cout<<"[ "<<segs_host[i]<<" , "<< segs_host[i+1] - 1 <<" ]"<<std::endl;
            }

            mgpu::mem_t<int> data = mgpu::to_mem(mem_gathered,context);
            mgpu::mem_t<int> segs = mgpu::to_mem(segs_host,context);
            mgpu::mem_t<int> values(edges_count, context);
            cout<<"initial gather"<< endl;
            for(int i = 0; i< edges_count;i++){
                std::cout<<dest_labels[i]<<" ";
            }
            cout<<endl;
            mgpu::segmented_sort(data.data(), values.data(), edges_count, segs.data(), vertices_count , mgpu::less_t<int>(), context);
            std::vector<int> values_host = from_mem(data); // gather
            std::cout<<"sorted gather"<<std::endl;

            for(int i = 0; i<values_host.size();i++){
                std::cout<<values_host[i]<<" ";
            }

            cout<<endl;

            {
                dim3 block(vertices_count, 1);
                dim3 grid(1, 1);

                SAFE_KERNEL_CALL((extract_boundaries_initial << < grid, block >> > (f_array, a.get_dev_v_array(), edges_count)));
            }
            {
                dim3 block(edges_count, 1);
                dim3 grid(1, 1);

                SAFE_KERNEL_CALL((extract_boundaries_optional << < grid, block >> > (f_array, data.data(), edges_count)));
            }

            cout<<"boundaries array"<<endl;
            bool* f_host = new bool[edges_count];
            hipMemcpy(f_host, f_array, (size_t )edges_count* sizeof(bool), hipMemcpyDeviceToHost);

            for(int i = 0; i< edges_count;i++){
                std::cout<<f_host[i]<<" ";
            }
            cout<<endl;

            std::vector<int> f_host1;
            for (int k = 0; k < edges_count; k++) {
               f_host1.push_back(f_host[k]);
            }

            mgpu::mem_t<int> f_mem = mgpu::to_mem(f_host1,context);

            mgpu::mem_t<int> output(edges_count, context);
            mgpu::scan(f_mem.data(),edges_count,output.data(),context);

            std::vector<int> hosted_bounds = from_mem(output); // gather

            std::cout<<"scanned f_array"<<std::endl;

            for(int i = 0; i< edges_count;i++){
                std::cout<<hosted_bounds[i]<<" ";
            }



            a.get_dev_v_array();
            a.get_dev_e_array();
            a.get_dev_weigths();
            SAFE_CALL(hipFree(dev_dest_labels));
            SAFE_CALL(hipFree(a.get_dev_v_array()));
            SAFE_CALL(hipFree(a.get_dev_e_array()));
            SAFE_CALL(hipFree(a.get_dev_weigths())); //check for unweigthed graph!
            SAFE_CALL(hipFree(f_array));



        }



        if (lp_flag) {
            lp(vertices_count, a.get_e_array(), a.get_v_array(), labels);
            //louvain(vertices_count, edges_count, a.get_e_array(), a.get_v_array(), labels,a.get_weights(),true);
            a.save_to_graphviz_file("graph_res", labels);
            label_stats(labels, vertices_count);
            delete[] labels;
        }
        delete[] src_ids;
        delete[] dst_ids;
        delete[] weights;

    }
    catch (const char *error) {
        cout << error << endl;
        getchar();
        return 1;
    }
    catch (...) {
        cout << "unknown error" << endl;
    }

    SAFE_CALL(hipProfilerStop());
    return 0;
}