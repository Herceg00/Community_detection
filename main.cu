#include "hip/hip_runtime.h"
#include "./moderngpu/kernel_segsort.hxx"
#include "./moderngpu/memory.hxx"
#include "./moderngpu/kernel_scan.hxx"
#include "./moderngpu/kernel_segreduce.hxx"

//#include "./moderngpu/src/moderngpu/kernel_segsort.hxx"
//#include "./moderngpu/src/moderngpu/memory.hxx"
//#include "./moderngpu/src/moderngpu/kernel_segreduce.hxx"
//#include "./moderngpu/src/moderngpu/kernel_scan.hxx"
//
//#include "/usr/local/cuda-10.1/include/hip/hip_runtime.h"
//#include "/usr/local/cuda-10.1/include/hip/hip_runtime_api.h"
#include <omp.h>
#include <iostream>
#include <string.h>
#include <fstream>
#include <algorithm>
#include "stdlib.h"
#include "CSR_GRAPH.h"
#include <stdio.h>
#include <math.h>
#include <sstream>
#include <string>
#include <vector>
#include "lp.h"
#include "CSR_GRAPH.h"
#include "generator.h"
#include "device_gather.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "map"



#define SAFE_CALL(CallInstruction) { \
    hipError_t cuerr = CallInstruction; \
    if(cuerr != hipSuccess) { \
         printf("CUDA error: %s at call \"" #CallInstruction "\"\n", hipGetErrorString(cuerr)); \
         throw "error in CUDA API function, aborting..."; \
    } \
}

#define SAFE_KERNEL_CALL(KernelCallInstruction){ \
    KernelCallInstruction; \
    hipError_t cuerr = hipGetLastError(); \
    if(cuerr != hipSuccess) { \
        printf("CUDA error in kernel launch: %s at kernel \"" #KernelCallInstruction "\"\n", hipGetErrorString(cuerr)); \
        throw "error in CUDA kernel launch, aborting..."; \
    } \
    cuerr = hipDeviceSynchronize(); \
    if(cuerr != hipSuccess) { \
        printf("CUDA error in kernel execution: %s at kernel \"" #KernelCallInstruction "\"\n", hipGetErrorString(cuerr)); \
        throw "error in CUDA kernel execution, aborting..."; \
    } \
}


void debug_info(std::vector<int> &ptr, string info) {
    cout << info << endl;
    for (int i = 0; i < ptr.size(); i++) {
        std::cout << ptr[i] << " ";
    }
    cout << endl;
}

void debug_info(int *ptr, int size_n, string info) {
    cout << info << endl;
    for (int i = 0; i < size_n; i++) {
        std::cout << ptr[i] << " ";
    }
    cout << endl;
}

void debug_info(unsigned int *ptr, int size_n, string info) {
    cout << info << endl;
    for (int i = 0; i < size_n; i++) {
        std::cout << ptr[i] << " ";
    }
    cout << endl;
}


void debug_info(bool *ptr, int size_n, string info) {
    cout << info << endl;
    for (int i = 0; i < size_n; i++) {
        std::cout << ptr[i] << " ";
    }
    cout << endl;
}


void print_bounds(std::vector<int> &ptr, int edges_count) {
    for (int i = 0; i < ptr.size(); i++) {
        if (i == 0) {
            //std::cout << "[ " << 0 << " , " << ptr[0] - 1 << " ]" << std::endl;
            std::cout << "[ " << ptr[i] << " , " << ptr[i + 1] - 1 << " ]" << std::endl;
            continue;
        }
        if (i == ptr.size() - 1) {
            std::cout << "[ " << ptr[ptr.size() - 1] << " , " << edges_count - 1 << " ]"
                      << std::endl;;
            continue;
        }
        std::cout << "[ " << ptr[i] << " , " << ptr[i + 1] - 1 << " ]" << std::endl;
    }
}


void label_stats(unsigned int *labels, unsigned int vertices_count) { // Почему то в map много нулей
    std::map<unsigned int, int> mp;
    for (unsigned int i = 0; i < vertices_count; i++) {
        if (mp.count(labels[i])) {
            mp[labels[i]]++;
        } else {
            mp[labels[i]] = 1;
        }
    }
    std::map<int, int> components;
    for (auto it = mp.begin(); it != mp.end(); it++) {
        if (components.count(it->second)) {
            components[it->second]++;
        } else {
            components[it->second] = 1;
        }
    }
    for (auto it = components.begin(); it != components.end(); it++) {
        if (it->first != 0) {
            cout << "there are " << it->second << " components of size " << it->first << endl;
        }
    }
}




void input(char *filename, bool directed, unsigned int *&src_ids, unsigned int *&dst_ids, unsigned int &vertices_count,
           unsigned int &edges_count) {
    unsigned int max_vertice = 0;

    std::ifstream infile(filename);
    std::string line;
    unsigned int i = 0;

    while (std::getline(infile, line)) {
        std::istringstream iss(line);
        int a, b;
        if (!(iss >> a >> b)) {
            break;
        } else {
            if (max(a, b) > max_vertice) {
                max_vertice = (unsigned) max(a, b);
            }
        }
        i++;
    }
    edges_count = i;
    vertices_count = max_vertice;
    src_ids = new unsigned int[edges_count];
    dst_ids = new unsigned int[edges_count];

    std::ifstream infile1(filename);
    i = 0;
    while (std::getline(infile1, line)) {
        std::istringstream iss(line);
        unsigned int a, b;
        if (!(iss >> a >> b)) {
            break;
        } else {
            src_ids[i] = a;
            dst_ids[i] = b;
            if (!directed) {
                src_ids[i + 1] = b;
                dst_ids[i + 1] = a;
            }
        }
        i++;
    }
}

__global__ void extract_boundaries_initial(short *boundaries, unsigned int *v_array, unsigned int edges_count) {

    unsigned long int i = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned long int position = v_array[i];
    if (i != 0) {
        boundaries[position - 1] = 1;
    } else {
        boundaries[edges_count - 1] = 1;
    }
}

__global__ void extract_boundaries_optional(short *boundaries, unsigned  int *dest_labels, unsigned int edges_count) {
    unsigned long int i = threadIdx.x + blockIdx.x * blockDim.x;
    if ((boundaries[i] != 1) && (i < edges_count)) {
        if (dest_labels[i] != dest_labels[i + 1]) {
            boundaries[i] = 1;
        }
    }
}

__global__ void count_labels(int *scanned_array, unsigned int edges_count, int *S_array) {
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    if ((i < edges_count - 1) && (scanned_array[i + 1] != scanned_array[i])) {
        S_array[scanned_array[i]] = i;
    }
}

__global__ void new_boundaries(int *scanned_array, unsigned int *v_array, unsigned int edges_count, int *S_ptr) {
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    S_ptr[i] = scanned_array[v_array[i]];
}

__global__ void frequency_count(int *W_array, int *S) {
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    if ((i > 0) && (S[i] != 0)) {
        W_array[i] = S[i] - S[i - 1];
    } else {
        W_array[0] = S[0] + 1;
    }
}

__global__ void get_labels(int *I , int* S, unsigned int *L ,unsigned int* labels){
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    labels[i] = L[S[I[i]]];
}


using namespace std;


int main(int argc, char **argv) {
    try {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        int threads = omp_get_max_threads();
        int vertices_index;
        int density_degree;
        bool check_flag = false;
        bool test_flag = false;
        char *graph_type;
        bool lp_flag = false;
        bool gather_flag = false;
        char *test_file = NULL;
        for (int i = 1; i < argc; i++) {
            string option(argv[i]);

            if ((option.compare("-scale") == 0) || (option.compare("-s") == 0)) {
                vertices_index = atoi(argv[++i]);
            }

            if ((option.compare("-edges") == 0) || (option.compare("-e") == 0)) {
                density_degree = atoi(argv[++i]);
            }

            if ((option.compare("-check") == 0)) {
                check_flag = true;
            }

            if ((option.compare("-nocheck") == 0)) {
                check_flag = false;
            }
            if ((option.compare("-type") == 0)) {
                graph_type = argv[++i];
            }
            if ((option.compare("-testing") == 0)) {
                test_file = argv[++i];
                test_flag = true;
                cout << "FLAG FOUND" << endl;
            }
            if ((option.compare("-lp")) == 0) {
                lp_flag = true;
            }
            if ((option.compare("-gather")) == 0) {
                gather_flag = true;
            }

        }

        unsigned int vertices_count = pow(2.0, vertices_index);
        unsigned int edges_count = density_degree * vertices_count;
        unsigned int *src_ids = NULL;
        unsigned int *dst_ids = NULL;
        float *weights = new float[edges_count];

        if (!test_flag) {
            src_ids = new unsigned int[edges_count];
            dst_ids = new unsigned int[edges_count];
            cout << "test_flag" << endl;
            if (strcmp(graph_type, "rmat") == 0) {
                R_MAT(src_ids, dst_ids, weights, vertices_count, edges_count, 45, 20, 20, 15, threads, true, true);

            } else {
                cout << "UR_GEN" << endl;
                uniform_random(src_ids, dst_ids, weights, vertices_count, edges_count, threads, true, true);
                cout << "Generated_UR" << endl;
            }
        } else {
            cout << test_flag << endl;
            cout << "file_init" << endl;
            input(test_file, true, src_ids, dst_ids, vertices_count, edges_count);
            vertices_count++;
            cout << "vertices:" << vertices_count << endl;
            cout << "edges: " << edges_count << endl;
        }


//        for (int i = 0; i < edges_count; i++) {
//            cout << src_ids[i] << "----" << dst_ids[i] << endl;
//        }

        cout << endl;
        CSR_GRAPH a(vertices_count, edges_count, src_ids, dst_ids, weights, true);
        a.save_to_graphviz_file("graph_pred", NULL);
        a.print_CSR_format();

        unsigned int *labels = new unsigned int[vertices_count];
        for (unsigned int j = 0; j < vertices_count; j++) {
            labels[j] = j;
        }
        cout << "LABELS : ";
        for (unsigned int j = 0; j < vertices_count; j++) {
            cout << labels[j] << " ";
        }
        unsigned int *dest_labels = new unsigned int[edges_count];
        unsigned int *dev_labels;
        unsigned int *dev_dest_labels;
        short *F_mem;

        if (gather_flag) {

            SAFE_CALL((hipMalloc((void **) &dev_labels, (size_t) (sizeof(unsigned int)) * (vertices_count))));
            SAFE_CALL((hipMalloc((void **) &dev_dest_labels, (size_t) (sizeof(unsigned int)) * edges_count)));
            SAFE_CALL((hipMalloc((void **) &F_mem, (size_t) (sizeof(short)) * edges_count)));

            a.move_to_device(dest_labels, labels, dev_dest_labels, dev_labels);

            int iter = 0;
            mgpu::standard_context_t context;
            do {
                SAFE_CALL(hipEventRecord(start));

                {
                    //Change configuration after
                    dim3 block(32 * vertices_count, 1);
                    dim3 grid(1, 1); //only for test
                    printf("starting...");
                    SAFE_KERNEL_CALL((gather_warp_per_vertex << < grid, block >> >
                                                                        (a.get_dev_v_array(), a.get_dev_e_array(), dev_dest_labels, dev_labels, edges_count, vertices_count)));
                }
                printf("terminating....");
                SAFE_CALL(hipEventRecord(stop));
                SAFE_CALL(hipEventSynchronize(stop));
                float time;
                SAFE_CALL(hipEventElapsedTime(&time, start, stop));
                time *= 1000000;
                //a.move_to_host(dest_labels, labels, dev_dest_labels, dev_labels);


                if (check_flag) {
                    unsigned int *test_dest_labels = new unsigned int[edges_count];
                    form_label_array(threads, vertices_count, edges_count, test_dest_labels, a.get_dev_v_array(),
                                     labels,
                                     a.get_e_array());
                    int flag = check(edges_count, dest_labels, test_dest_labels);
                    if (flag == 0) {
                        printf("CORRECT");
                    }
                    delete[] test_dest_labels;
                }


                printf("GATHER Bandwidth for 2^%d vertices and 2^%d edges is %f GB/s\n ", vertices_index,
                       vertices_index + (int) log2((double) density_degree),
                       sizeof(unsigned int) * (2 * vertices_count + 2 * edges_count) / (time));


                //mgpu::standard_context_t context;

//                std::vector<int> L; //GATHER
//                for (int k = 0; k < edges_count; k++) {
//                    L.push_back(dest_labels[k]);
//                }

                std::vector<int> ptr; //Bounds as segments
                for (int k = 0; k < vertices_count; k++) {
                    ptr.push_back(a.get_v_array()[k]);
                }

                print_bounds(ptr, edges_count);

//                mgpu::mem_t<int> data = mgpu::to_mem(L, context);
                mgpu::mem_t<int> segs = mgpu::to_mem(ptr, context);
                mgpu::mem_t<int> values(edges_count, context);

                //debug_info(L, "initial gather");


                mgpu::segmented_sort(dev_dest_labels, values.data(), edges_count, segs.data(), vertices_count,
                                     mgpu::less_t<int>(), context);

//                std::vector<int> L_sorted = from_mem(data); // gather
//
//                debug_info(L_sorted, "sorted_gather");

                SAFE_CALL((hipMemset(F_mem, 0, (size_t) (sizeof(short)) * edges_count))); //was taken from group of memcpy

                {
                    dim3 block(vertices_count, 1);
                    dim3 grid(1, 1);

                    SAFE_KERNEL_CALL(
                            (extract_boundaries_initial << < grid, block >> >
                                                                   (F_mem, a.get_dev_v_array(), edges_count))); //fill 1 in bounds
                }
                {
                    dim3 block(edges_count, 1);
                    dim3 grid(1, 1);

                    SAFE_KERNEL_CALL(
                            (extract_boundaries_optional << < grid, block >> >
                                                                    (F_mem, dev_dest_labels, edges_count))); //sub(i+1, i)
                }

//                bool *F_host = new bool[edges_count];
//                hipMemcpy(F_host, F_mem, (size_t) edges_count * sizeof(bool), hipMemcpyDeviceToHost);
//
//                debug_info(F_host, edges_count, "neighbors");

//                std::vector<int> F_host_;
//                for (int k = 0; k < edges_count; k++) {
//                    F_host_.push_back(F_host[k]);
//                }

                //mgpu::mem_t<int> F_memt = mgpu::to_mem(F_host_, context);


                mgpu::mem_t<int> F_scanned(edges_count, context);

                mgpu::scan(F_mem, edges_count, F_scanned.data(), context); // may not work because of bool

//                std::vector<int> hosted_bounds = from_mem(F_scanned); // gather
//
//                debug_info(hosted_bounds, "scanned F");

                int reduced_size;
                hipMemcpy(&reduced_size, &F_scanned.data()[edges_count - 1], sizeof(int), hipMemcpyDeviceToHost);

                mgpu::mem_t<int> s_array(reduced_size, context);

                {
                    dim3 block(edges_count, 1);
                    dim3 grid(1, 1);
                    SAFE_KERNEL_CALL(
                            (count_labels << < grid, block >> > (F_scanned.data(), edges_count, s_array.data())));
                }


//                std::vector<int> s_host = from_mem(s_array);
//
//                debug_info(s_host, "S for frequency");

                mgpu::mem_t<int> s_ptr_array(vertices_count, context);

                {
                    dim3 block(vertices_count, 1);
                    dim3 grid(1, 1);
                    SAFE_KERNEL_CALL((new_boundaries << < grid, block >> >
                                                                (F_scanned.data(), a.get_dev_v_array(), edges_count, s_ptr_array.data())));
                }

//                std::vector<int> ptr_host = from_mem(s_ptr_array);
//
//                debug_info(ptr_host, "new bounds");

                mgpu::mem_t<int> w_array(reduced_size, context);
                {
                    dim3 block(reduced_size, 1);
                    dim3 grid(1, 1);


                    SAFE_KERNEL_CALL((frequency_count << < grid, block >> > (w_array.data(), s_array.data())));
                }

//                std::vector<int> w_host = from_mem(w_array);
//                std::vector<int> debug_w((size_t) reduced_size);
//                debug_info(w_host, "W_array");

                mgpu::mem_t<int> out(vertices_count, context);

                std::vector<int> I;
                for (int k = 0; k < reduced_size; k++) { //indices for reduce
                    I.push_back(k);
                }

                mgpu::mem_t<int> I_mem = mgpu::to_mem(I, context);

                int init = 0;


//            auto k = [] MGPU_DEVICE(int tid, int cta) {
//
//            };
                int *w_ptr = w_array.data();

                auto my_cool_lambda =[w_ptr] MGPU_DEVICE(int
                a, int
                b) ->int{
                        if ( w_ptr[a] > w_ptr[b]){
                            return a;
                        } else{
                            return b;
                        }
                };


                mgpu::segreduce(I_mem.data(), reduced_size, s_ptr_array.data(), vertices_count, out.data(),
                                my_cool_lambda, (int) init, context);

//                std::vector<int> i_host = from_mem(out);
//                debug_info(i_host, "seg_reduce");


                {
                    dim3 block(vertices_count, 1);
                    dim3 grid(1, 1);
                    SAFE_KERNEL_CALL((get_labels << < grid, block >> >
                                                            (out.data(), s_array.data(), dev_dest_labels, dev_labels)));
                }

                hipMemcpy(labels,dev_labels,vertices_count,hipMemcpyDeviceToHost);
                std::cout<<"Iteration "<<iter<< " is over"<<endl;
                debug_info(labels,vertices_count,"Labels after current iteration");

                iter++;
            }while (iter <4);
            a.get_dev_v_array();
            a.get_dev_e_array();
            a.get_dev_weigths();
            SAFE_CALL(hipFree(dev_dest_labels));
            SAFE_CALL(hipFree(dev_labels));
            SAFE_CALL(hipFree(a.get_dev_v_array()));
            SAFE_CALL(hipFree(a.get_dev_e_array()));
            SAFE_CALL(hipFree(a.get_dev_weigths())); //check for unweigthed graph!
            SAFE_CALL(hipFree(F_mem));


        }


        if (lp_flag) {
            lp(vertices_count, a.get_e_array(), a.get_v_array(), labels);
            //louvain(vertices_count, edges_count, a.get_e_array(), a.get_v_array(), labels,a.get_weights(),true);
            a.save_to_graphviz_file("graph_res", labels);
            label_stats(labels, vertices_count);
            delete[] labels;
        }
        delete[] src_ids;
        delete[] dst_ids;
        delete[] weights;

    }
    catch (const char *error) {
        cout << error << endl;
        getchar();
        return 1;
    }
    catch (...) {
        cout << "unknown error" << endl;
    }

    SAFE_CALL(hipProfilerStop());
    return 0;
}