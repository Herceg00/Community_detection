#include "hip/hip_runtime.h"
//#include "./moderngpu/src/moderngpu/kernel_segsort.hxx"
//#include "./moderngpu/src/moderngpu/memory.hxx"
#include "./moderngpu/kernel_segsort.hxx"
#include "./moderngpu/memory.hxx"
#include <omp.h>
#include <iostream>
#include <string.h>
#include <fstream>
#include <algorithm>
#include "stdlib.h"
#include "CSR_GRAPH.h"
#include <stdio.h>
#include <math.h>
#include <sstream>
#include <string>
#include <vector>
#include "lp.h"
#include "CSR_GRAPH.h"
#include "generator.h"

//#include "./moderngpu/kernel_segsort.hxx"
//#include "./moderngpu/memory.hxx"
#include "device_gather.h"

//#include "/usr/local/cuda-10.1/include/hip/hip_runtime.h"
#include "hip/hip_runtime.h"
//#include "/usr/local/cuda-10.1/include/hip/hip_runtime_api.h"
#include "hip/hip_runtime_api.h"

#include "map"

#define SAFE_CALL(CallInstruction) { \
    hipError_t cuerr = CallInstruction; \
    if(cuerr != hipSuccess) { \
         printf("CUDA error: %s at call \"" #CallInstruction "\"\n", hipGetErrorString(cuerr)); \
         throw "error in CUDA API function, aborting..."; \
    } \
}

#define SAFE_KERNEL_CALL(KernelCallInstruction){ \
    KernelCallInstruction; \
    hipError_t cuerr = hipGetLastError(); \
    if(cuerr != hipSuccess) { \
        printf("CUDA error in kernel launch: %s at kernel \"" #KernelCallInstruction "\"\n", hipGetErrorString(cuerr)); \
        throw "error in CUDA kernel launch, aborting..."; \
    } \
    cuerr = hipDeviceSynchronize(); \
    if(cuerr != hipSuccess) { \
        printf("CUDA error in kernel execution: %s at kernel \"" #KernelCallInstruction "\"\n", hipGetErrorString(cuerr)); \
        throw "error in CUDA kernel execution, aborting..."; \
    } \
}


void label_stats(unsigned int *labels, unsigned int vertices_count) { // Почему то в map много нулей
    std::map<unsigned int, int> mp;
    for (unsigned int i = 0; i < vertices_count; i++) {
        if (mp.count(labels[i])) {
            mp[labels[i]]++;
        } else {
            mp[labels[i]] = 1;
        }
    }
    std::map<int, int> components;
    for (auto it = mp.begin(); it != mp.end(); it++) {
        if (components.count(it->second)) {
            components[it->second]++;
        } else {
            components[it->second] = 1;
        }
    }
    for (auto it = components.begin(); it != components.end(); it++) {
        if (it->first != 0) {
            cout << "there are " << it->second << " components of size " << it->first << endl;
        }
    }
}



void input(char *filename, bool directed, unsigned int *&src_ids, unsigned int *&dst_ids, unsigned int &vertices_count,
           unsigned int &edges_count) {
    unsigned int max_vertice = 0;

    std::ifstream infile(filename);
    std::string line;
    unsigned int i = 0;

    while (std::getline(infile, line)) {
        std::istringstream iss(line);
        int a, b;
        if (!(iss >> a >> b)) {
            break;
        } else {
            if (max(a, b) > max_vertice) {
                max_vertice = (unsigned) max(a, b);
            }
        }
        i++;
    }
    edges_count = i;
    vertices_count = max_vertice;
    src_ids = new unsigned int[edges_count];
    dst_ids = new unsigned int[edges_count];

    std::ifstream infile1(filename);
    i = 0;
    while (std::getline(infile1, line)) {
        std::istringstream iss(line);
        unsigned int a, b;
        if (!(iss >> a >> b)) {
            break;
        } else {
            src_ids[i] = a;
            dst_ids[i] = b;
            if (!directed) {
                src_ids[i + 1] = b;
                dst_ids[i + 1] = a;
            }
        }
        i++;
    }
}


using namespace std;


int main(int argc, char **argv) {
    try {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        int threads = omp_get_max_threads();
        int vertices_index;
        int density_degree;
        bool check_flag = false;
        bool test_flag = false;
        char *graph_type;
        bool lp_flag = false;
        bool gather_flag = false;
        char *test_file = NULL;
        for (int i = 1; i < argc; i++) {
            string option(argv[i]);

            if ((option.compare("-scale") == 0) || (option.compare("-s") == 0)) {
                vertices_index = atoi(argv[++i]);
            }

            if ((option.compare("-edges") == 0) || (option.compare("-e") == 0)) {
                density_degree = atoi(argv[++i]);
            }

            if ((option.compare("-check") == 0)) {
                check_flag = true;
            }

            if ((option.compare("-nocheck") == 0)) {
                check_flag = false;
            }
            if ((option.compare("-type") == 0)) {
                graph_type = argv[++i];
            }
            if ((option.compare("-testing") == 0)) {
                test_file = argv[++i];
                test_flag = true;
                cout << "FLAG FOUND" << endl;
            }
            if ((option.compare("-lp")) == 0) {
                lp_flag = true;
            }
            if ((option.compare("-gather")) == 0) {
                gather_flag = true;
            }

        }

        unsigned int vertices_count = pow(2.0, vertices_index);
        unsigned int edges_count = density_degree * vertices_count;
        unsigned int *src_ids = NULL;
        unsigned int *dst_ids = NULL;
        float *weights = new float[edges_count];

        if (!test_flag) {
            src_ids = new unsigned int[edges_count];
            dst_ids = new unsigned int[edges_count];
            cout << "test_flag" << endl;
            if (strcmp(graph_type, "rmat") == 0) {
                R_MAT(src_ids, dst_ids, weights, vertices_count, edges_count, 45, 20, 20, 15, threads, true, true);

            } else {
                cout << "UR_GEN" << endl;
                uniform_random(src_ids, dst_ids, weights, vertices_count, edges_count, threads, true, true);
                cout << "Generated_UR" << endl;
            }
        } else {
            cout << test_flag << endl;
            cout << "file_init" << endl;
            input(test_file, false, src_ids, dst_ids, vertices_count, edges_count);
            vertices_count++;
            cout << "vertices:" << vertices_count << endl;
            cout << "edges: " << edges_count << endl;
        }


//        for (int i = 0; i < edges_count; i++) {
//            cout << src_ids[i] << "----" << dst_ids[i] << endl;
//        }

        cout << endl;
        CSR_GRAPH a(vertices_count, edges_count, src_ids, dst_ids, weights, true);
        a.save_to_graphviz_file("graph_pred", NULL);
        a.print_CSR_format();

        unsigned int *labels = new unsigned int[vertices_count];
        unsigned int *dest_labels = new unsigned int[edges_count];
        unsigned int *dev_labels;
        unsigned int *dev_dest_labels;

        if (gather_flag) {

            SAFE_CALL((hipMalloc((void **) &dev_labels, (size_t) (sizeof(unsigned int)) * (vertices_count))));
            SAFE_CALL((hipMalloc((void **) &dev_dest_labels, (size_t) (sizeof(unsigned int)) * edges_count)));

            a.move_to_device(dest_labels, labels, dev_dest_labels, dev_labels);

            SAFE_CALL(hipEventRecord(start));
            //dim3 block(1024, 1);
            //dim3 grid(vertices_count * 32 / block.x, 1);
            dim3 block(vertices_count * 32 , 1);
            dim3 grid(1,1);

            printf("starting...");
            SAFE_KERNEL_CALL((gather_warp_per_vertex <<< grid, block >>>
                                                                (a.get_dev_v_array(), a.get_dev_e_array(), dev_dest_labels, dev_labels, edges_count, vertices_count)));
            printf("terminating....");
            SAFE_CALL(hipEventRecord(stop));
            SAFE_CALL(hipEventSynchronize(stop));
            float time;
            SAFE_CALL(hipEventElapsedTime(&time, start, stop));
            time *= 1000000;
            a.move_to_host(dest_labels, labels, dev_dest_labels, dev_labels);
            SAFE_CALL(hipFree(dev_labels));
            SAFE_CALL(hipFree(dev_dest_labels));

            if (check_flag) {
                unsigned int *test_dest_labels = new unsigned int[edges_count];
                form_label_array(threads, vertices_count, edges_count, test_dest_labels, a.get_dev_v_array(), labels,
                                 a.get_e_array());
                int flag = check(edges_count, dest_labels, test_dest_labels);
                if (flag == 0) {
                    printf("CORRECT");
                }
                delete[] test_dest_labels;
            }


            printf("GATHER Bandwidth for 2^%d vertices and 2^%d edges is %f GB/s\n ", vertices_index,
                   vertices_index + (int) log2((double) density_degree),
                   sizeof(unsigned int) * (2 * vertices_count + 2 * edges_count) / (time));

            mgpu::standard_context_t context;
            std::vector<unsigned int> mem_gathered;
            for (int k = 0; k < edges_count; k++) {
                mem_gathered.push_back(dest_labels[k]);
            }
            std::vector<unsigned int> segs_host;
            for (int k = 0; k < vertices_count; k++) {
                segs_host.push_back(a.get_v_array()[k]);
            }
            
            for(int i = 0; i<segs_host.size();i++){
                if(i == 0){
                    std::cout<<"[ "<<0<<" , "<<segs_host[0] - 1<<" ]"<<std::endl;
                    std::cout<<"[ "<<segs_host[i]<<" , "<< segs_host[i+1] - 1 <<" ]"<<std::endl;
                    continue;
                }
                if(i == segs_host.size() - 1){
                    std::cout<<"[ "<<segs_host[segs_host.size() - 1 ]<<" , "<<  edges_count -1 <<" ]"<<std::endl; ;
                    continue;
                }
                std::cout<<"[ "<<segs_host[i]<<" , "<< segs_host[i+1] - 1 <<" ]"<<std::endl;
            }
            mgpu::mem_t<unsigned int> data = mgpu::to_mem(mem_gathered,context);
            mgpu::mem_t<unsigned int> segs = mgpu::to_mem(segs_host,context);
            mgpu::mem_t<unsigned int> values(edges_count, context);

            mgpu::segmented_sort(data.data(), values.data(), edges_count, segs.data(), vertices_count , mgpu::less_t<int>(), context);
            std::vector<int> values_host = from_mem(data);

            for(int i = 0; i<values_host.size();i++){
                std::cout<<i<<" -th element is "<<values_host[i]<<" "<<std::endl;
            }
        }
        //cout<<"2"<<endl;
        if (lp_flag) {
            lp(vertices_count, a.get_e_array(), a.get_v_array(), labels);
            a.save_to_graphviz_file("graph_res", labels);
            label_stats(labels, vertices_count);
            delete[] labels;
        }
        delete[] src_ids;
        delete[] dst_ids;
        delete[] weights;

    }
    catch (const char *error) {
        cout << error << endl;
        getchar();
        return 1;
    }
    catch (...) {
        cout << "unknown error" << endl;
    }

    SAFE_CALL(hipProfilerStop());
    return 0;
}