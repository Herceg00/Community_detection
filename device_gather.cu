#include "hip/hip_runtime.h"


//#include "/usr/local/cuda-10.1/include/hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "device_gather.h"
#include <iostream>
#include <stdio.h>
#include <assert.h>

#define SAFE_CALL( CallInstruction ) { \
    hipError_t cuerr = CallInstruction; \
    if(cuerr != hipSuccess) { \
         printf("CUDA error: %s at call \"" #CallInstruction "\"\n", hipGetErrorString(cuerr)); \
		 throw "error in CUDA API function, aborting..."; \
    } \
}

#define SAFE_KERNEL_CALL( KernelCallInstruction ){ \
    KernelCallInstruction; \
    hipError_t cuerr = hipGetLastError(); \
    if(cuerr != hipSuccess) { \
        printf("CUDA error in kernel launch: %s at kernel \"" #KernelCallInstruction "\"\n", hipGetErrorString(cuerr)); \
		throw "error in CUDA kernel launch, aborting..."; \
    } \
    cuerr = hipDeviceSynchronize(); \
    if(cuerr != hipSuccess) { \
        printf("CUDA error in kernel execution: %s at kernel \"" #KernelCallInstruction "\"\n", hipGetErrorString(cuerr)); \
		throw "error in CUDA kernel execution, aborting..."; \
    } \
}



__global__ void device_gather(unsigned int *v_array,unsigned int *e_array,unsigned int *dest_labels ,unsigned int *labels,
                              unsigned long  long edges, unsigned long long vertices) {

    unsigned int i = threadIdx.x + blockIdx.x*blockDim.x;

    assert(i < vertices+1);

    int v_begin = v_array[i];
    int v_end = v_array[i+1];


    for (int j = v_begin; j < v_end; ++j) {
        assert(e_array[j] < edges);
        dest_labels[j] = labels[e_array[j]];
    }
}



void CSR_GRAPH::move_to_device(void) {

    SAFE_CALL((hipMalloc((unsigned**)&dev_v_array,(size_t)sizeof(this->dev_v_array[0])*(vertices_count+1))));
    SAFE_CALL((hipMalloc((unsigned**)&dev_e_array,(size_t)sizeof(this->dev_e_array[0])*edges_count)));
    if(weighted){
        SAFE_CALL((hipMalloc((float**)&dev_weigths,(size_t)sizeof(this->dev_e_array[0])*edges_count)));
    }
    SAFE_CALL((hipMalloc((unsigned**)&dev_labels,(size_t)sizeof(this->dev_v_array[0])*(vertices_count+1))));
    SAFE_CALL((hipMalloc((unsigned**)&dev_dest_labels,(size_t)sizeof(this->dev_e_array[0])*edges_count)));


    SAFE_CALL((hipMemcpy(dev_dest_labels,dest_labels,(size_t)sizeof(this->dev_e_array[0])*edges_count,hipMemcpyHostToDevice)));
    SAFE_CALL((hipMemcpy(dev_v_array,v_array,(size_t)(vertices_count+1)*(this->dev_v_array[0]),hipMemcpyHostToDevice)));
    SAFE_CALL((hipMemcpy(dev_e_array,e_array,(size_t)sizeof(this->dev_e_array[0])*edges_count,hipMemcpyHostToDevice)));
    SAFE_CALL((hipMemcpy(dev_weigths,weigths,(size_t)sizeof(this->dev_e_array[0])*edges_count,hipMemcpyHostToDevice)));
    SAFE_CALL((hipMemcpy(dev_labels,labels,(size_t)sizeof(this->dev_v_array[0])*(vertices_count+1),hipMemcpyHostToDevice)));
    std::cout<<"moved to device"<<std::endl;

}


void CSR_GRAPH::move_to_host (void) {

    SAFE_CALL((hipMemcpy(v_array,dev_v_array,(size_t)(vertices_count+1)* sizeof(this->dev_v_array[0]),hipMemcpyDeviceToHost)));
    SAFE_CALL((hipMemcpy(e_array,dev_e_array,(size_t)edges_count* sizeof(this->dev_e_array[0]),hipMemcpyDeviceToHost)));
    SAFE_CALL((hipMemcpy(weigths,dev_weigths,(size_t)edges_count* sizeof(this->dev_e_array[0]),hipMemcpyDeviceToHost)));
    SAFE_CALL((hipMemcpy(labels,dev_labels,(size_t)(vertices_count+1)* sizeof(this->dev_v_array[0]),hipMemcpyDeviceToHost)));
    SAFE_CALL((hipMemcpy(dest_labels,dev_dest_labels,(size_t)edges_count* sizeof(this->dev_e_array[0]),hipMemcpyDeviceToHost)));

    hipFree(dev_v_array);
    hipFree(dev_e_array);
    if(weighted){
        hipFree(dev_weigths);
    }
    hipFree(dev_labels);
    hipFree(dev_dest_labels);
    std::cout<<"moved back"<<std::endl;

}
