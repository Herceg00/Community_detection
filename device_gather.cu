#include "hip/hip_runtime.h"


//#include "/usr/local/cuda-10.1/include/hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "device_gather.h"
#include <iostream>
#include <stdio.h>




__global__ void device_gather(unsigned int *v_array,unsigned int *e_array,unsigned int *dest_labels ,unsigned int *labels) {

    unsigned int i = threadIdx.x + blockIdx.x*blockDim.x;
    int v_begin = v_array[i];
    int v_end = v_array[i+1];

    for (int j = v_begin; j < v_end; ++j) {
        dest_labels[j] = labels[e_array[j]];
    }
}



void CSR_GRAPH::move_to_device(void) {
    hipMalloc((unsigned**)&dev_v_array,(size_t)vertices_count);
    hipMalloc((unsigned**)&dev_e_array,(size_t)edges_count);
    if(weighted){
        hipMalloc((float**)&dev_weigths,(size_t)edges_count);
    }
    hipMalloc((unsigned**)&dev_labels,(size_t)edges_count);
    hipMalloc((unsigned**)&dev_dest_labels,(size_t)edges_count);

    hipMemcpy(dev_dest_labels,dest_labels,(size_t)edges_count,hipMemcpyHostToDevice);
    hipMemcpy(dev_v_array,v_array,(size_t)vertices_count,hipMemcpyHostToDevice);
    hipMemcpy(dev_e_array,e_array,(size_t)edges_count,hipMemcpyHostToDevice);
    hipMemcpy(dev_weigths,weigths,(size_t)edges_count,hipMemcpyHostToDevice);
    hipMemcpy(dev_labels,labels,(size_t)vertices_count,hipMemcpyHostToDevice);
    std::cout<<"moved to device"<<std::endl;

}


void CSR_GRAPH::move_to_host (void) {

    hipMemcpy(v_array,dev_v_array,(size_t)vertices_count,hipMemcpyDeviceToHost);
    hipMemcpy(e_array,dev_e_array,(size_t)edges_count,hipMemcpyDeviceToHost);
    hipMemcpy(weigths,dev_weigths,(size_t)edges_count,hipMemcpyDeviceToHost);
    hipMemcpy(labels,dev_labels,(size_t)vertices_count,hipMemcpyDeviceToHost);
    hipMemcpy(dest_labels,dev_dest_labels,(size_t)edges_count,hipMemcpyDeviceToHost);

    hipFree(dev_v_array);
    hipFree(dev_e_array);
    if(weighted){
        hipFree(dev_weigths);
    }
    hipFree(dev_labels);
    hipFree(dev_dest_labels);
    std::cout<<"moved back"<<std::endl;

}
