#include "hip/hip_runtime.h"


//#include "/usr/local/cuda-10.1/include/hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "device_gather.h"
#include <iostream>
#include <stdio.h>
#include <assert.h>
#define WARP_SIZE 32

#define SAFE_CALL( CallInstruction ) { \
    hipError_t cuerr = CallInstruction; \
    if(cuerr != hipSuccess) { \
         printf("CUDA error: %s at call \"" #CallInstruction "\"\n", hipGetErrorString(cuerr)); \
		 throw "error in CUDA API function, aborting..."; \
    } \
}

#define SAFE_KERNEL_CALL( KernelCallInstruction ){ \
    KernelCallInstruction; \
    hipError_t cuerr = hipGetLastError(); \
    if(cuerr != hipSuccess) { \
        printf("CUDA error in kernel launch: %s at kernel \"" #KernelCallInstruction "\"\n", hipGetErrorString(cuerr)); \
		throw "error in CUDA kernel launch, aborting..."; \
    } \
    cuerr = hipDeviceSynchronize(); \
    if(cuerr != hipSuccess) { \
        printf("CUDA error in kernel execution: %s at kernel \"" #KernelCallInstruction "\"\n", hipGetErrorString(cuerr)); \
		throw "error in CUDA kernel execution, aborting..."; \
    } \
}



__global__ void gather_warp_per_vertex(  int *v_array,  int *e_array,  int *dest_labels ,  int *labels,
                                         long  long edges,   long long vertices) {

      long int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < vertices*WARP_SIZE) {

          long long warp_num = i / 32 ;
        short warp_pos = i % 32;
          long long int v_begin = v_array[warp_num];
          long long int v_end = v_array[warp_num + 1];
          long long position = v_begin + warp_pos;
        while(position < v_end){
            dest_labels[position] = labels[e_array[position]];
            position+=WARP_SIZE;
        }
    }
}






void CSR_GRAPH::move_to_device(  int* dest_labels,   int* labels,   int* dev_dest_labels ,  int* dev_labels) {

    SAFE_CALL((hipMalloc((void**)&dev_v_array,(size_t)sizeof(this->v_array[0])*(vertices_count+1))));
    SAFE_CALL((hipMalloc((void**)&dev_e_array,(size_t)sizeof(this->e_array[0])*edges_count)));
    if(weighted){
        SAFE_CALL((hipMalloc((void**)&dev_weigths,(size_t)sizeof(this->e_array[0])*edges_count)));
    }


    SAFE_CALL((hipMemcpy(dev_dest_labels,dest_labels,(size_t)(sizeof(this->e_array[0])*edges_count),hipMemcpyHostToDevice)));
    SAFE_CALL((hipMemcpy(dev_v_array,v_array,(size_t)((vertices_count+1)* sizeof(this->v_array[0])),hipMemcpyHostToDevice)));
    SAFE_CALL((hipMemcpy(dev_e_array,e_array,(size_t)(sizeof(this->e_array[0])*edges_count),hipMemcpyHostToDevice)));
    SAFE_CALL((hipMemcpy(dev_weigths,weights,(size_t)(sizeof(this->e_array[0])*edges_count),hipMemcpyHostToDevice)));
    SAFE_CALL((hipMemcpy(dev_labels,labels,(size_t)(sizeof(this->v_array[0])*(vertices_count)),hipMemcpyHostToDevice)));

    std::cout<<"moved to device"<<std::endl;

}


void CSR_GRAPH::move_to_host (  int* dest_labels,   int* labels,   int* dev_dest_labels ,  int* dev_labels) {

    SAFE_CALL((hipMemcpy(v_array,dev_v_array,(size_t)(vertices_count+1)* sizeof(this->v_array[0]),hipMemcpyDeviceToHost)));
    SAFE_CALL((hipMemcpy(e_array,dev_e_array,(size_t)edges_count* sizeof(this->e_array[0]),hipMemcpyDeviceToHost)));
    SAFE_CALL((hipMemcpy(weights,dev_weigths,(size_t)edges_count* sizeof(this->e_array[0]),hipMemcpyDeviceToHost)));
    SAFE_CALL((hipMemcpy(labels,dev_labels,(size_t)(vertices_count)* sizeof(this->v_array[0]),hipMemcpyDeviceToHost)));
    SAFE_CALL((hipMemcpy(dest_labels,dev_dest_labels,(size_t)edges_count* sizeof(this->e_array[0]),hipMemcpyDeviceToHost)));

    //SAFE_CALL(hipFree(dev_v_array));
    //SAFE_CALL(hipFree(dev_e_array));
    //if(weighted){
    //    SAFE_CALL(hipFree(dev_weigths));
    //}

    std::cout<<"moved back"<<std::endl;

}