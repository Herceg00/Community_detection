#include "hip/hip_runtime.h"


//#include "/usr/local/cuda-10.1/include/hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "device_gather.h"





__global__ void device_gather(unsigned int *v_array,unsigned int *e_array,unsigned int *dest_labels ,unsigned int *labels) {

    unsigned int i = threadIdx.x + blockIdx.x*blockDim.x;

    int v_begin = v_array[i];
    int v_end = v_array[i+1];

    for (int j = v_begin; j < v_end; ++j) {
        dest_labels[j] = labels[e_array[j]];
    }
}

void CSR_GRAPH::move_to_device(void) {


    hipMalloc((unsigned**)&dev_v_array,(size_t)vertices_count);
    hipMalloc((unsigned**)&dev_e_array,(size_t)edges_count);
    if(weighted){
        hipMalloc((float**)&dev_weigths,(size_t)edges_count);
    }
    hipMalloc((unsigned**)&dev_labels,(size_t)edges_count);
    hipMalloc((unsigned**)&dev_dest_labels,(size_t)edges_count);

    hipMemcpy(dev_v_array,v_array,(size_t)vertices_count,hipMemcpyHostToDevice);
    hipMemcpy(dev_e_array,e_array,(size_t)edges_count,hipMemcpyHostToDevice);
    hipMemcpy(dev_weigths,weigths,(size_t)edges_count,hipMemcpyHostToDevice);
    hipMemcpy(dev_labels,labels,(size_t)vertices_count,hipMemcpyHostToDevice);
}


void CSR_GRAPH::move_to_host (void) {

    hipFree(dev_v_array);
    hipFree(dev_e_array);
    if(weighted){
        hipFree(dev_weigths);
    }

    hipFree(dev_labels);
    hipFree(dev_dest_labels);

    hipMemcpy(v_array,dev_v_array,(size_t)vertices_count,hipMemcpyDeviceToHost);
    hipMemcpy(e_array,dev_e_array,(size_t)edges_count,hipMemcpyDeviceToHost);
    hipMemcpy(weigths,dev_weigths,(size_t)edges_count,hipMemcpyDeviceToHost);
    hipMemcpy(labels,dev_labels,(size_t)vertices_count,hipMemcpyDeviceToHost);
    hipMemcpy(dest_labels,dev_dest_labels,(size_t)edges_count,hipMemcpyDeviceToHost);
}
